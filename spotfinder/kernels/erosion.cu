#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "../spotfinder.h"
#include "erosion.hu"

namespace cg = cooperative_groups;

#pragma region Device Functions
/**
 * @brief Load central pixels into shared memory.
 * @param block The cooperative group for the current block.
 * @param mask Pointer to the mask data.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param radius The radius around each masked pixel to be considered.
 */
__device__ void load_central_pixels(cg::thread_block block,
                                    const uint8_t *mask,
                                    uint8_t *shared_mask,
                                    size_t mask_pitch,
                                    int width,
                                    int height,
                                    int radius) {
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;
    int local_x = block.thread_index().x + radius;
    int local_y = block.thread_index().y + radius;
    int shared_width = block.group_dim().x + 2 * radius;

    // Load central pixels into shared memory
    if (x < width && y < height) {
        shared_mask[local_y * shared_width + local_x] = mask[y * mask_pitch + x];
    } else {
        shared_mask[local_y * shared_width + local_x] = MASKED_PIXEL;
    }
}

/**
 * @brief Load border pixels into shared memory.
 * @param block The cooperative group for the current block.
 * @param mask Pointer to the mask data.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param radius The radius around each masked pixel to be considered.
 */
__device__ void load_border_pixels(cg::thread_block block,
                                   const uint8_t *mask,
                                   uint8_t *shared_mask,
                                   size_t mask_pitch,
                                   int width,
                                   int height,
                                   int radius) {
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;
    int local_x = block.thread_index().x + radius;
    int local_y = block.thread_index().y + radius;
    int shared_width = block.group_dim().x + 2 * radius;
    int shared_height = block.group_dim().y + 2 * radius;

    // Load border pixels into shared memory
    for (int i = block.thread_index().x; i < shared_width; i += block.group_dim().x) {
        for (int j = block.thread_index().y; j < shared_height;
             j += block.group_dim().y) {
            int global_x = x + (i - local_x);
            int global_y = y + (j - local_y);

            bool is_within_central_region =
              (i >= radius && i < shared_width - radius && j >= radius
               && j < shared_height - radius);
            bool is_global_x_in_bounds = (global_x >= 0 && global_x < width);
            bool is_global_y_in_bounds = (global_y >= 0 && global_y < height);

            if (is_within_central_region) {
                continue;
            }

            if (is_global_x_in_bounds && is_global_y_in_bounds) {
                shared_mask[j * shared_width + i] =
                  mask[global_y * mask_pitch + global_x];
            } else {
                shared_mask[j * shared_width + i] = MASKED_PIXEL;
            }
        }
    }
}

/**
 * @brief Determine if the current pixel should be erased based on the mask.
 * @param block The cooperative group for the current block.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param radius The radius around each masked pixel to be considered.
 * @param distance_threshold The maximum Chebyshev distance for erasing the current pixel.
 * @return True if the current pixel should be erased, false otherwise.
 */
__device__ bool determine_erasure(cg::thread_block block,
                                  const uint8_t *shared_mask,
                                  int radius,
                                  int distance_threshold) {
    int local_x = block.thread_index().x + radius;
    int local_y = block.thread_index().y + radius;
    int shared_width = block.group_dim().x + 2 * radius;

    bool should_erase = false;
    for (int i = -radius; i <= radius; ++i) {
        for (int j = -radius; j <= radius; ++j) {
            if (shared_mask[(local_y + j) * shared_width + (local_x + i)]
                == MASKED_PIXEL) {
                int chebyshev_distance = max(abs(i), abs(j));
                if (chebyshev_distance <= distance_threshold) {
                    should_erase = true;
                    break;
                }
            }
        }
        if (should_erase) {
            break;
        }
    }
    return should_erase;
}

// __global__ void determine_erasure_kernel(const uint8_t *shared_mask,
//                                          int shared_width,
//                                          int local_x,
//                                          int local_y,
//                                          int radius,
//                                          int distance_threshold,
//                                          unsigned int *should_erase) {
//     int i = threadIdx.x - radius;
//     int j = threadIdx.y - radius;

//     if (shared_mask[(local_y + j) * shared_width + (local_x + i)] == MASKED_PIXEL) {
//         int chebyshev_distance = max(abs(i), abs(j));
//         if (chebyshev_distance <= distance_threshold) {
//             atomicExch(should_erase, 1u);
//         }
//     }
// }

/**
 * @brief Device function to determine if the current pixel should be erased using dynamic parallelism.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param threadParams Thread-specific information for the current thread.
 * @param radius The radius around each masked pixel to be considered.
 * @param distance_threshold The maximum Chebyshev distance for erasing the current pixel.
 * @return True if the current pixel should be erased, false otherwise.
 */
// __device__ bool launch_determine_erasure_kernel(const uint8_t *shared_mask,
//                                                 const KernelThreadParams &threadParams,
//                                                 int radius,
//                                                 int distance_threshold) {
//     // Allocate memory for the erasure flag
//     unsigned int *d_should_erase;
//     hipMalloc(&d_should_erase, sizeof(unsigned int));
//     hipMemset(d_should_erase, 0, sizeof(unsigned int));

//     // Launch the erasure determination kernel
//     dim3 erasure_block_size(2 * radius + 1, 2 * radius + 1);
//     determine_erasure_kernel<<<1, erasure_block_size>>>(shared_mask,
//                                                         threadParams.shared_width,
//                                                         threadParams.local_x,
//                                                         threadParams.local_y,
//                                                         radius,
//                                                         distance_threshold,
//                                                         d_should_erase);

//     // Copy the result back to the host
//     unsigned int h_should_erase_uint;
//     hipMemcpy(&h_should_erase_uint,
//                d_should_erase,
//                sizeof(unsigned int),
//                hipMemcpyDeviceToHost);
//     hipFree(d_should_erase);

//     return h_should_erase_uint == 1u;
// }
#pragma endregion Device Functions

#pragma region Kernel
/**
 * @brief CUDA kernel to apply erosion based on the mask and update the erosion_mask.
 * 
 * This kernel uses shared memory to store a local copy of the mask for each block.
 * 
 * @param mask Pointer to the mask data indicating valid pixels.
 * @param erosion_mask Pointer to the allocated output erosion mask.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param erosion_mask Pointer to the output erosion mask data. (Expected to be the same size as the mask)
 * @param width The width of the image.
 * @param height The height of the image.
 * @param radius The radius around each masked pixel to also be masked.
 */
__global__ void erosion_kernel(
  const uint8_t __restrict__ *mask,
  uint8_t __restrict__ *erosion_mask,
  // __restrict__ is a hint to the compiler that the two pointers are not
  // aliased, allowing the compiler to perform more agressive optimizations
  size_t mask_pitch,
  int width,
  int height,
  int radius) {
    // Declare shared memory to store a local copy of the mask for the block
    extern __shared__ uint8_t shared_mask[];

    // Create a cooperative group for the current block
    cg::thread_block block = cg::this_thread_block();

    // Load central pixels
    load_central_pixels(block, mask, shared_mask, mask_pitch, width, height, radius);

    // Load border pixels
    load_border_pixels(block, mask, shared_mask, mask_pitch, width, height, radius);

    // Synchronize threads to ensure all shared memory is loaded
    block.sync();

    /*
     * If the current pixel is outside the image bounds, return without doing anything.
     * We do this after loading shared memory as it may be necessary for this thread 
     * to load border pixels.
    */
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    if (x >= width || y >= height) return;
    if (mask[y * mask_pitch + x] != VALID_PIXEL) {
        erosion_mask[y * mask_pitch + x] = VALID_PIXEL;
        return;
    }

    // Determine if the current pixel should be erased
    bool should_erase = determine_erasure(
      block, shared_mask, radius, 2);  // Use 2 as the Chebyshev distance threshold

    // dynamic parrelism based
    // bool should_erase_gpu =
    //   launch_determine_erasure_kernel(shared_mask,
    //                                   threadParams,
    //                                   radius,
    //                                   2);  // Use 2 as the Chebyshev distance threshold

    // Update the erosion_mask based on erosion result
    if (should_erase) {
        erosion_mask[y * mask_pitch + x] = MASKED_PIXEL;
    } else {
        erosion_mask[y * mask_pitch + x] = VALID_PIXEL;
    }
}
#pragma endregion Kernel