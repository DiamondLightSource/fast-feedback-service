#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "../spotfinder.cuh"
#include "erosion.cuh"

// Macro to get the value of a pitched array
#define GET_PITCHED_VALUE(array, pitch, x, y) (array[y * pitch + x])

namespace cg = cooperative_groups;

#pragma region Erosion kernel
__global__ void erosion_kernel(
  uint8_t __restrict__ *dispersion_mask,
  uint8_t __restrict__ *erosion_mask,
  uint8_t __restrict__ *mask,
  // __restrict__ is a hint to the compiler that the two pointers are not
  // aliased, allowing the compiler to perform more agressive optimizations
  size_t dispersion_mask_pitch,
  size_t erosion_mask_pitch,
  size_t mask_pitch,
  int width,
  int height,
  uint8_t radius) {
    // Calculate the pixel coordinates
    auto block = cg::this_thread_block();
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    // Guards
    if (x >= width || y >= height) return;  // Out of bounds guard
    bool is_background = dispersion_mask[y * dispersion_mask_pitch + x] == 0;
    if (is_background) {
        /*
         * If the pixel is masked, we want to set it to VALID_PIXEL
         * in order to invert the mask. However, this cannot be done
         * before the erosion step is complete, as the mask is used
         * to determine whether to erode. Instead, this thread skips
         * the unecessary processing and does this at the end.
        */
        erosion_mask[y * erosion_mask_pitch + x] = 1;
        return;
    }
    // Calculate the bounds of the erosion kernel
    int x_start = max(0, x - radius);
    int x_end = min(x + radius + 1, width);
    int y_start = max(0, y - radius);
    int y_end = min(y + radius + 1, height);

    bool should_erase = false;
    constexpr uint8_t chebyshev_distance_threshold = 2;

    // Iterate over the kernel bounds
    for (int kernel_x = x_start; kernel_x < x_end; ++kernel_x) {
        for (int kernel_y = y_start; kernel_y < y_end; ++kernel_y) {
            /*
             * TODO: Investigate whether we should be doing this or not!
            */
            // if (mask[kernel_y * mask_pitch + kernel_x] == 0) {
            //     continue;
            // }
            if (dispersion_mask[kernel_y * dispersion_mask_pitch + kernel_x] == 0) {
                // If the current pixel is background, check the Chebyshev distance
                uint8_t chebyshev_distance = max(abs(kernel_x - x), abs(kernel_y - y));

                if (chebyshev_distance <= chebyshev_distance_threshold) {
                    // If a background pixel is too close, the current pixel should be erased
                    should_erase = true;
                    // We can then break out of the loop, as no further checks are necessary
                    goto termination;
                }
            }
        }
    }

termination:
    if (should_erase) {
        erosion_mask[y * erosion_mask_pitch + x] = 1;
    } else {
        erosion_mask[y * erosion_mask_pitch + x] =
          !dispersion_mask[y * dispersion_mask_pitch + x];
    }
}
#pragma enregion Erosion kernel