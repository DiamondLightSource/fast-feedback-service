#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "../spotfinder.cuh"
#include "erosion.cuh"

namespace cg = cooperative_groups;

#pragma region Device Functions
/**
 * @brief Load central pixels into shared memory.
 * @param block The cooperative group for the current block.
 * @param mask Pointer to the mask data.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param radius The radius around each masked pixel to be considered.
 */
__device__ void load_central_pixels(cg::thread_block block,
                                    const uint8_t *mask,
                                    uint8_t *shared_mask,
                                    size_t mask_pitch,
                                    int width,
                                    int height,
                                    int radius) {
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;
    int local_x = block.thread_index().x + radius;
    int local_y = block.thread_index().y + radius;
    int shared_width = block.group_dim().x + 2 * radius;

    // Load central pixels into shared memory
    if (x < width && y < height) {
        shared_mask[local_y * shared_width + local_x] = mask[y * mask_pitch + x];
    } else {
        shared_mask[local_y * shared_width + local_x] = VALID_PIXEL;
    }
}

/**
 * @brief Load border pixels into shared memory.
 * @param block The cooperative group for the current block.
 * @param mask Pointer to the mask data.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param radius The radius around each masked pixel to be considered.
 */
__device__ void load_border_pixels(cg::thread_block block,
                                   const uint8_t *mask,
                                   uint8_t *shared_mask,
                                   size_t mask_pitch,
                                   int width,
                                   int height,
                                   int radius) {
    // Calculate the global x and y coordinates for the current thread
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    // Calculate the x and y coordinates in shared memory, including the border
    int local_x = block.thread_index().x + radius;
    int local_y = block.thread_index().y + radius;

    // Calculate the width and height of the shared memory buffer, including the border
    int shared_width = block.group_dim().x + 2 * radius;
    int shared_height = block.group_dim().y + 2 * radius;

    // Load top and bottom borders.
    if (block.thread_index().y < radius) {  // If the thread is in the top border region
        // Top border: Load pixels from the rows above the central block region into shared memory.
        // Use max(y - radius, 0) to ensure that we don't read outside the image boundary.
        int border_y = max(y - radius, 0);
        shared_mask[(block.thread_index().y) * shared_width + local_x] =
          mask[border_y * mask_pitch + x];

        // Bottom border: Load pixels from the rows below the central block region into shared memory.
        // Use min(y + block.group_dim().y, height - 1) to ensure that we don't read beyond the image.
        border_y = min(y + block.group_dim().y, height - 1);
        shared_mask[(local_y + block.group_dim().y) * shared_width + local_x] =
          mask[border_y * mask_pitch + x];
    }

    // Load left and right borders.
    if (block.thread_index().x
        < radius) {  // If the thread is in the left border region
        // Left border: Load pixels from columns to the left of the central block region into shared memory.
        // Use max(x - radius, 0) to ensure that we don't read outside the image boundary.
        int border_x = max(x - radius, 0);
        shared_mask[local_y * shared_width + block.thread_index().x] =
          mask[y * mask_pitch + border_x];

        // Right border: Load pixels from columns to the right of the central block region into shared memory.
        // Use min(x + block.group_dim().x, width - 1) to ensure that we don't read beyond the image.
        border_x = min(x + block.group_dim().x, width - 1);
        shared_mask[local_y * shared_width + (local_x + block.group_dim().x)] =
          mask[y * mask_pitch + border_x];
    }

    // Load corner pixels to fill in the gaps at the corners of the shared memory region.
    // This ensures that the entire shared memory area surrounding the central region is populated.
    if (block.thread_index().x < radius && block.thread_index().y < radius) {
        // Top-left corner: Load the pixel at the intersection of the top row and left column.
        int border_x = max(x - radius, 0);
        int border_y = max(y - radius, 0);
        shared_mask[block.thread_index().y * shared_width + block.thread_index().x] =
          mask[border_y * mask_pitch + border_x];

        // Top-right corner: Load the pixel at the intersection of the top row and right column.
        border_x = min(x + block.group_dim().x, width - 1);
        shared_mask[block.thread_index().y * shared_width
                    + (local_x + block.group_dim().x)] =
          mask[border_y * mask_pitch + border_x];

        // Bottom-left corner: Load the pixel at the intersection of the bottom row and left column.
        border_y = min(y + block.group_dim().y, height - 1);
        shared_mask[(local_y + block.group_dim().y) * shared_width
                    + block.thread_index().x] = mask[border_y * mask_pitch + border_x];

        // Bottom-right corner: Load the pixel at the intersection of the bottom row and right column.
        shared_mask[(local_y + block.group_dim().y) * shared_width
                    + (local_x + block.group_dim().x)] =
          mask[border_y * mask_pitch + border_x];
    }
}

/**
 * @brief Determine if the current pixel should be erased based on the mask.
 * @param block The cooperative group for the current block.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param radius The radius around each masked pixel to be considered.
 * @param distance_threshold The maximum Chebyshev distance for erasing the current pixel.
 * @return True if the current pixel should be erased, false otherwise.
 */
__device__ bool determine_erasure(cg::thread_block block,
                                  const uint8_t *shared_mask,
                                  int radius,
                                  int distance_threshold) {
    int local_x = block.thread_index().x + radius;
    int local_y = block.thread_index().y + radius;
    int shared_width = block.group_dim().x + 2 * radius;

    bool should_erase = false;
    for (int i = -radius; i <= radius; ++i) {
        for (int j = -radius; j <= radius; ++j) {
            if (shared_mask[(local_y + j) * shared_width + (local_x + i)]
                == MASKED_PIXEL) {
                int chebyshev_distance = max(abs(i), abs(j));
                if (chebyshev_distance <= distance_threshold) {
                    should_erase = true;
                    break;
                }
            }
        }
        if (should_erase) {
            break;
        }
    }
    return should_erase;
}

// __global__ void determine_erasure_kernel(const uint8_t *shared_mask,
//                                          int shared_width,
//                                          int local_x,
//                                          int local_y,
//                                          int radius,
//                                          int distance_threshold,
//                                          unsigned int *should_erase) {
//     int i = threadIdx.x - radius;
//     int j = threadIdx.y - radius;

//     if (shared_mask[(local_y + j) * shared_width + (local_x + i)] == MASKED_PIXEL) {
//         int chebyshev_distance = max(abs(i), abs(j));
//         if (chebyshev_distance <= distance_threshold) {
//             atomicExch(should_erase, 1u);
//         }
//     }
// }

/**
 * @brief Device function to determine if the current pixel should be erased using dynamic parallelism.
 * @param shared_mask Pointer to the shared memory buffer.
 * @param threadParams Thread-specific information for the current thread.
 * @param radius The radius around each masked pixel to be considered.
 * @param distance_threshold The maximum Chebyshev distance for erasing the current pixel.
 * @return True if the current pixel should be erased, false otherwise.
 */
// __device__ bool launch_determine_erasure_kernel(const uint8_t *shared_mask,
//                                                 const KernelThreadParams &threadParams,
//                                                 int radius,
//                                                 int distance_threshold) {
//     // Allocate memory for the erasure flag
//     unsigned int *d_should_erase;
//     hipMalloc(&d_should_erase, sizeof(unsigned int));
//     hipMemset(d_should_erase, 0, sizeof(unsigned int));

//     // Launch the erasure determination kernel
//     dim3 erasure_block_size(2 * radius + 1, 2 * radius + 1);
//     determine_erasure_kernel<<<1, erasure_block_size>>>(shared_mask,
//                                                         threadParams.shared_width,
//                                                         threadParams.local_x,
//                                                         threadParams.local_y,
//                                                         radius,
//                                                         distance_threshold,
//                                                         d_should_erase);

//     // Copy the result back to the host
//     unsigned int h_should_erase_uint;
//     hipMemcpy(&h_should_erase_uint,
//                d_should_erase,
//                sizeof(unsigned int),
//                hipMemcpyDeviceToHost);
//     hipFree(d_should_erase);

//     return h_should_erase_uint == 1u;
// }
#pragma endregion Device Functions

#pragma region Erosion kernel(s)
/**
 * @brief CUDA kernel to apply erosion based on the mask and update the erosion_mask.
 * 
 * This kernel uses shared memory to store a local copy of the mask for each block.
 * 
 * @param mask Pointer to the mask data indicating valid pixels to be eroded.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param radius The radius around each masked pixel to also be masked.
 */
__global__ void erosion_kernel(
  uint8_t __restrict__ *mask,
  // __restrict__ is a hint to the compiler that the two pointers are not
  // aliased, allowing the compiler to perform more agressive optimizations
  size_t mask_pitch,
  int width,
  int height,
  uint8_t radius) {
    // Declare shared memory to store a local copy of the mask for the block
    extern __shared__ uint8_t shared_mask[];

    // Create a cooperative group for the current block
    cg::thread_block block = cg::this_thread_block();

    // Load central pixels
    load_central_pixels(block, mask, shared_mask, mask_pitch, width, height, radius);

    // Load border pixels
    load_border_pixels(block, mask, shared_mask, mask_pitch, width, height, radius);

    // Synchronize threads to ensure all shared memory is loaded
    block.sync();

    /*
     * If the current pixel is outside the image bounds, return without doing anything.
     * We do this after loading shared memory as it may be necessary for this thread 
     * to load border pixels.
    */
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    if (x == 0 && y == 0) {
        printf("group dim x: %d\n", block.group_dim().x);
        printf("group dim y: %d\n", block.group_dim().y);
    }

    // Return if the current pixel is outside the image bounds
    if (x >= width || y >= height) return;

    /*
     * If the current pixel is not a signal pixel, mark it as valid and return.
     * We do not need to perform erosion on non-signal pixels, but we need them
     * to be marked as valid in order to allow the background calculation to proceed.
    */
    if (mask[y * mask_pitch + x] == 0) {
        mask[y * mask_pitch + x] = VALID_PIXEL;
        return;
    }

    constexpr uint8_t chebyshev_distance_threshold = 2;

    // Determine if the current pixel should be erased
    bool should_erase =
      determine_erasure(block, shared_mask, radius, chebyshev_distance_threshold);
    // DIALS uses 2 as the Chebyshev distance threshold for erasing pixels

    // dynamic parrelism based
    // bool should_erase_gpu =
    //   launch_determine_erasure_kernel(shared_mask,
    //                                   threadParams,
    //                                   radius,
    //                                   2);  // Use 2 as the Chebyshev distance threshold

    // Update the erosion_mask based on erosion result
    if (should_erase) {
        /*
         * Erase the pixel from the background mask. This is done by setting the pixel
         * as valid (i.e. not masked) in the mask data. This allows the pixel to be
         * considered as a background pixel in the background calculation as it is not
         * considered part of the signal.
        */
        mask[y * mask_pitch + x] = VALID_PIXEL;
    } else {
        /*
         * If the pixel should not be erased, this means that it is part of the signal.
         * and needs to be marked as masked in the mask data. This prevents the pixel
         * from being considered as part of the background in the background calculation.
        */

        // Invert 'valid' signal spot to 'masked' background spots
        mask[y * mask_pitch + x] = !mask[y * mask_pitch + x];
    }
}
#pragma endregion Kernel