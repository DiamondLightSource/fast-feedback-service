#include "hip/hip_runtime.h"
/**
 * Basic Naive Kernel
 * 
 * Does spotfinding in-kernel, without in-depth performance tweaking.
 * 
 */

// #include <bitshuffle.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <lodepng.h>

#include "kernels/erosion.cuh"
#include "kernels/thresholding.cuh"
#include "spotfinder.cuh"

namespace cg = cooperative_groups;

#pragma region Res Mask Functions
/**
 * @brief Function to calculate the distance of a pixel from the beam center.
 * @param x The x-coordinate of the pixel in the image
 * @param y The y-coordinate of the pixel in the image
 * @param center_x The x-coordinate of the pixel beam center in the image
 * @param center_y The y-coordinate of the pixel beam center in the image
 * @param pixel_size_x The pixel size of the detector in the x-direction in m
 * @param pixel_size_y The pixel size of the detector in the y-direction in m
 * @return The calculated distance from the beam center in m
*/
__device__ float get_distance_from_center(float x,
                                          float y,
                                          float center_x,
                                          float center_y,
                                          float pixel_size_x,
                                          float pixel_size_y) {
    /*
     * Since this calculation is for a broad, general exclusion, we can
     * use basic Pythagoras to calculate the distance from the center.
     * 
     * We add 0.5 in order to move to the center of the pixel. Since
     * starting at 0, 0, for instance, would infact be the corner.
    */
    float dx = ((x + 0.5f) - center_x) * pixel_size_x;
    float dy = ((y + 0.5f) - center_y) * pixel_size_y;

    return sqrtf(dx * dx + dy * dy);
}

/**
 * @brief Function to calculate the interplanar distance of a reflection.
 * The interplanar distance is calculated using the formula:
 *         d = λ / (2 * sin(ϴ))
 * @param wavelength The wavelength of the X-ray beam in Å
 * @param distance_to_detector The distance from the sample to the detector in m
 * @param distance_from_center The distance of the reflection from the beam center in m
 * @return The calculated d value
*/
__device__ float get_resolution(float wavelength,
                                float distance_to_detector,
                                float distance_from_center) {
    /*
     * Since the angle calculated is, in fact, 2ϴ, we halve to get the
     * proper value of ϴ
    */
    float theta = 0.5 * atanf(distance_from_center / distance_to_detector);
    return wavelength / (2 * sinf(theta));
}
#pragma endregion Res Mask Functions

#pragma region Res Mask Kernel
/**
 * @brief CUDA kernel to apply a resolution mask for an image.
 *
 * This kernel calculates the resolution for each pixel in an image based on the
 * distance from the beam center and the detector properties. It then masks out
 * pixels whose resolution falls outside the specified range [dmin, dmax],
 * provided that the pixel is not already masked, by setting the mask value of
 * the pixel to 0 in the mask data.
 *
 * @param mask Pointer to the mask data indicating valid pixels.
 * @param mask_pitch The pitch of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param wavelength The wavelength of the X-ray beam in Ångströms.
 * @param distance_to_detector The distance from the sample to the detector in m.
 * @param beam_center_x The x-coordinate of the beam center in the image.
 * @param beam_center_y The y-coordinate of the beam center in the image.
 * @param pixel_size_x The pixel size of the detector in the x-direction in m.
 * @param pixel_size_y The pixel size of the detector in the y-direction in m.
 * @param dmin The minimum resolution (d-spacing) threshold.
 * @param dmax The maximum resolution (d-spacing) threshold.
 */
__global__ void apply_resolution_mask(uint8_t *mask,
                                      size_t mask_pitch,
                                      int width,
                                      int height,
                                      float wavelength,
                                      float distance_to_detector,
                                      float beam_center_x,
                                      float beam_center_y,
                                      float pixel_size_x,
                                      float pixel_size_y,
                                      float dmin,
                                      float dmax) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > width || y > height) return;  // Out of bounds

    if (mask[y * mask_pitch + x] == MASKED_PIXEL) {  // Check if the pixel is masked
        /*
        * If the pixel is already masked, we don't need to calculate the
        * resolution for it, so we can just leave it masked
        */
        return;
    }

    float distance_from_center = get_distance_from_center(
      x, y, beam_center_x, beam_center_y, pixel_size_x, pixel_size_y);
    float resolution =
      get_resolution(wavelength, distance_to_detector, distance_from_center);

    // Check if dmin is set and if the resolution is below it
    if (dmin > 0 && resolution < dmin) {
        mask[y * mask_pitch + x] = MASKED_PIXEL;
        return;
    }

    // Check if dmax is set and if the resolution is above it
    if (dmax > 0 && resolution > dmax) {
        mask[y * mask_pitch + x] = MASKED_PIXEL;
        return;
    }

    // If the pixel is not masked and the resolution is within the limits, set the resolution mask to 1
    mask[y * mask_pitch + x] = VALID_PIXEL;
    // ⛔🧊
}

/**
 * @brief Host function to launch the apply_resolution_mask kernel.
 *
 * This function sets up the kernel execution parameters and launches the
 * apply_resolution_mask kernel to generate and apply a resolution mask
 * onto the base mask for the detector.
 *
 * @param blocks The dimensions of the grid of blocks.
 * @param threads The dimensions of the grid of threads within each block.
 * @param shared_memory The size of shared memory required per block (in bytes).
 * @param stream The CUDA stream to execute the kernel.
 * @param mask Device pointer to the mask data indicating valid pixels.
 * @param params The parameters required to calculate the resolution mask.  
 */
void call_apply_resolution_mask(dim3 blocks,
                                dim3 threads,
                                size_t shared_memory,
                                hipStream_t stream,
                                uint8_t *mask,
                                ResolutionMaskParams params) {
    // Launch the kernel
    apply_resolution_mask<<<blocks, threads, shared_memory, stream>>>(
      mask,
      params.mask_pitch,
      params.width,
      params.height,
      params.wavelength,
      params.detector.distance,
      params.detector.beam_center_x,
      params.detector.beam_center_y,
      params.detector.pixel_size_x,
      params.detector.pixel_size_y,
      params.dmin,
      params.dmax);
}
#pragma endregion Res Mask Kernel

#pragma region Spotfinding Functions
/**
 * @brief Calculate the sum, sum of squares, and count of valid pixels in the neighborhood.
 * @param image Device pointer to the image data.
 * @param mask Device pointer to the mask data indicating valid pixels.
 * @param background_mask (Optional) Device pointer to the background mask data. If nullptr, all pixels are considered for background calculation.
 * @param image_pitch The pitch of the image data.
 * @param mask_pitch The pitch of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param x The x-coordinate of the current pixel.
 * @param y The y-coordinate of the current pixel.
 * @param kernel_width The half-width of the kernel (kernel size in x-direction).
 * @param kernel_height The half-height of the kernel (kernel size in y-direction).
 * @param sum (Output) The sum of the valid pixels in the neighborhood.
 * @param sumsq (Output) The sum of squares of the valid pixels in the neighborhood.
 * @param n (Output) The count of valid pixels in the neighborhood.
 */
__device__ void calculate_sums(pixel_t *image,
                               uint8_t *mask,
                               uint8_t *background_mask,
                               size_t image_pitch,
                               size_t mask_pitch,
                               int width,
                               int height,
                               int x,
                               int y,
                               uint8_t kernel_width,
                               uint8_t kernel_height,
                               uint &sum,
                               size_t &sumsq,
                               uint8_t &n) {
    sum = 0;
    sumsq = 0;
    n = 0;

    for (int row = max(0, y - kernel_height); row < min(y + kernel_height + 1, height);
         ++row) {
        int row_offset = image_pitch * row;
        int mask_offset = mask_pitch * row;
        for (int col = max(0, x - kernel_width); col < min(x + kernel_width + 1, width);
             ++col) {
            pixel_t pixel = image[row_offset + col];
            uint8_t mask_pixel = mask[mask_offset + col];
            bool include_pixel = mask_pixel != 0;  // If the pixel is valid
            if (background_mask != nullptr) {
                uint8_t background_mask_pixel = background_mask[mask_offset + col];
                include_pixel =
                  include_pixel
                  && (background_mask_pixel
                      == VALID_PIXEL);  // And is NOT a survivor from the erosion process
            }
            if (include_pixel) {
                sum += pixel;
                sumsq += pixel * pixel;
                n += 1;
            }
        }
    }
}

/**
 * @brief Determine if the current pixel is a strong pixel.
 * @param sum The sum of the valid pixels in the neighborhood.
 * @param sumsq The sum of squares of the valid pixels in the neighborhood.
 * @param n The count of valid pixels in the neighborhood.
 * @param this_pixel The intensity value of the current pixel.
 * @return True if the current pixel is a strong pixel, false otherwise.
 */
__device__ bool is_strong_pixel(uint sum, size_t sumsq, uint8_t n, pixel_t this_pixel) {
    constexpr float n_sig_s = 3.0f;
    constexpr float n_sig_b = 6.0f;

    float sum_f = static_cast<float>(sum);
    float sumsq_f = static_cast<float>(sumsq);

    float mean = sum_f / n;
    float variance = (n * sumsq_f - (sum_f * sum_f)) / (n * (n - 1));
    float dispersion = variance / mean;
    float background_threshold = 1 + n_sig_b * sqrt(2.0f / (n - 1));
    bool not_background = dispersion > background_threshold;
    float signal_threshold = mean + n_sig_s * sqrt(mean);
    bool is_signal = this_pixel > signal_threshold;

    return not_background && is_signal;
}
#pragma endregion Spotfinding Functions

#pragma region Spotfinding Kernels
/**
 * @brief CUDA kernel to perform spotfinding using a dispersion-based algorithm.
 * 
 * This kernel identifies strong pixels in the image based on analysis of the pixel neighborhood.
 * 
 * @param image Device pointer to the image data.
 * @param image_pitch The pitch of the image data.
 * @param mask Device pointer to the mask data indicating valid pixels.
 * @param background_mask (Optional) Device pointer to the background mask data. If nullptr, all pixels are considered for background calculation.
 * @param mask_pitch The pitch of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param kernel_width The radius of the kernel in the x-direction.
 * @param kernel_height The radius of the kernel in the y-direction.
 * @param result_strong (Output) Device pointer for the strong pixel mask data to be written to.
 */
__global__ void do_spotfinding_dispersion(pixel_t *image,
                                          size_t image_pitch,
                                          uint8_t *mask,
                                          uint8_t *background_mask,
                                          size_t mask_pitch,
                                          int width,
                                          int height,
                                          pixel_t max_valid_pixel_value,
                                          uint8_t kernel_width,
                                          uint8_t kernel_height,
                                          uint8_t *result_strong) {
    image = image + (image_pitch * height * blockIdx.z);
    // result_sum = result_sum + (image_pitch * height * blockIdx.z);
    // result_sumsq = result_sumsq + (image_pitch * height * blockIdx.z);
    // result_n = result_n + (mask_pitch * height * blockIdx.z);
    result_strong = result_strong + (mask_pitch * height * blockIdx.z);

    auto block = cg::this_thread_block();
    // auto warp = cg::tiled_partition<32>(block);
    // int warpId = warp.meta_group_rank();
    // int lane = warp.thread_rank();

    uint sum = 0;
    size_t sumsq = 0;
    uint8_t n = 0;

    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    // Don't calculate for masked pixels
    pixel_t this_pixel = image[y * image_pitch + x];
    bool px_is_valid =
      mask[y * mask_pitch + x] != 0 && this_pixel <= max_valid_pixel_value;

    if (px_is_valid) {
        calculate_sums(image,
                       mask,
                       background_mask,
                       image_pitch,
                       mask_pitch,
                       width,
                       height,
                       x,
                       y,
                       kernel_width,
                       kernel_height,
                       sum,
                       sumsq,
                       n);
    }

    if (x < width && y < height) {
        // result_sum[x + image_pitch * y] = sum;
        // result_sumsq[x + image_pitch * y] = sumsq;
        // result_n[x + mask_pitch * y] = n;

        // Calculate the thresholding
        if (px_is_valid && n > 1) {
            bool is_strong_pixel_flag = is_strong_pixel(sum, sumsq, n, this_pixel);
            result_strong[x + mask_pitch * y] = is_strong_pixel_flag;
        } else {
            result_strong[x + mask_pitch * y] = 0;
        }
    }
}
#pragma endregion Spotfinding Kernel

#pragma region Launch Wrappers
/**
 * @brief Wrapper function to call the dispersion-based spotfinding algorithm.
 * This function launches the `compute_dispersion_threshold_kernel` to perform
 * the spotfinding based on the basic dispersion threshold.
 *
 * @param blocks The dimensions of the grid of blocks.
 * @param threads The dimensions of the grid of threads within each block.
 * @param shared_memory The size of shared memory required per block (in bytes).
 * @param stream The CUDA stream to execute the kernel.
 * @param image PitchedMalloc object for the image data.
 * @param mask PitchedMalloc object for the mask data indicating valid pixels.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param result_strong (Output) Device pointer for the strong pixel mask data to be written to.
 * @param min_count The minimum number of valid pixels required in the local neighborhood. Default is 3.
 * @param n_sig_b The background noise significance level. Default is 6.0.
 * @param n_sig_s The signal significance level. Default is 3.0.
 */
void call_do_spotfinding_dispersion(dim3 blocks,
                                    dim3 threads,
                                    size_t shared_memory,
                                    hipStream_t stream,
                                    PitchedMalloc<pixel_t> &image,
                                    PitchedMalloc<uint8_t> &mask,
                                    int width,
                                    int height,
                                    pixel_t max_valid_pixel_value,
                                    PitchedMalloc<uint8_t> *result_strong,
                                    uint8_t min_count,
                                    float n_sig_b,
                                    float n_sig_s) {
    /// One-direction width of kernel. Total kernel span is (K * 2 + 1)
    constexpr uint8_t basic_kernel_radius = 3;

    // Launch the dispersion threshold kernel
    compute_threshold_kernel<<<blocks, threads, shared_memory, stream>>>(
      image.get(),            // Image data pointer
      mask.get(),             // Mask data pointer
      result_strong->get(),   // Output mask pointer
      image.pitch,            // Image pitch
      mask.pitch,             // Mask pitch
      result_strong->pitch,   // Output mask pitch
      width,                  // Image width
      height,                 // Image height
      max_valid_pixel_value,  // Maximum valid pixel value
      basic_kernel_radius,    // Kernel width
      basic_kernel_radius,    // Kernel height
      min_count,              // Minimum count
      n_sig_b,                // Background significance level
      n_sig_s                 // Signal significance level
    );

    // do_spotfinding_dispersion<<<blocks, threads, shared_memory, stream>>>(
    //   image.get(),
    //   image.pitch,
    //   mask.get(),
    //   nullptr,  // No background mask
    //   mask.pitch,
    //   width,
    //   height,
    //   max_valid_pixel_value,
    //   basic_kernel_radius,
    //   basic_kernel_radius,
    //   result_strong->get());

    hipStreamSynchronize(
      stream);  // Synchronize the CUDA stream to ensure the kernel is complete
}

/**
 * @brief Wrapper function to call the extended dispersion-based spotfinding algorithm.
 * This function launches the `compute_final_threshold_kernel` for final thresholding
 * after applying the dispersion mask and the `compute_dispersion_threshold_kernel`
 * for initial thresholding.
 *
 * @param blocks The dimensions of the grid of blocks.
 * @param threads The dimensions of the grid of threads within each block.
 * @param shared_memory The size of shared memory required per block (in bytes).
 * @param stream The CUDA stream to execute the kernel.
 * @param image PitchedMalloc object for the image data.
 * @param mask PitchedMalloc object for the mask data indicating valid pixels.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param result_strong (Output) Device pointer for the strong pixel mask data to be written to.
 * @param do_writeout Flag to indicate if the output should be written to file. Default is false.
 * @param min_count The minimum number of valid pixels required in the local neighborhood. Default is 3.
 * @param n_sig_b The background noise significance level. Default is 6.0.
 * @param n_sig_s The signal significance level. Default is 3.0.
 * @param threshold The global threshold for intensity values. Default is 10.0.
 */
void call_do_spotfinding_extended(dim3 blocks,
                                  dim3 threads,
                                  size_t shared_memory,
                                  hipStream_t stream,
                                  PitchedMalloc<pixel_t> &image,
                                  PitchedMalloc<uint8_t> &mask,
                                  int width,
                                  int height,
                                  pixel_t max_valid_pixel_value,
                                  PitchedMalloc<uint8_t> *result_strong,
                                  bool do_writeout,
                                  uint8_t min_count,
                                  float n_sig_b,
                                  float n_sig_s,
                                  float threshold) {
    // Allocate intermediate buffer for the dispersion mask on the device
    PitchedMalloc<uint8_t> d_dispersion_mask(width, height);

    constexpr uint8_t first_pass_kernel_radius = 3;

    /*
     * First pass
     * Perform the initial dispersion thresholding only on the background
     * threshold. The surviving pixels are then used as a mask later to
     * exclude them from the background calculation in the second pass.
    */
    {
        printf("First pass\n");
        // First pass: Perform the initial dispersion thresholding
        compute_dispersion_threshold_kernel<<<blocks, threads, shared_memory, stream>>>(
          image.get(),               // Image data pointer
          mask.get(),                // Mask data pointer
          d_dispersion_mask.get(),   // Output dispersion mask pointer
          image.pitch,               // Image pitch
          mask.pitch,                // Mask pitch
          d_dispersion_mask.pitch,   // Output dispersion mask pitch
          width,                     // Image width
          height,                    // Image height
          max_valid_pixel_value,     // Maximum valid pixel value
          first_pass_kernel_radius,  // Kernel radius
          first_pass_kernel_radius,  // Kernel radius
          min_count,                 // Minimum count
          n_sig_b,                   // Background significance level
          n_sig_s                    // Signal significance level
        );
        hipStreamSynchronize(
          stream);  // Synchronize the CUDA stream to ensure the first pass is complete

        printf("First pass complete\n");
        // Optional: Write out the first pass result if needed
        if (do_writeout) {
            // Write to PNG
            {
                // Function to transform the pixel values: if non-zero, set to 255, otherwise set to 0
                auto convert_pixel = [](uint8_t pixel) -> uint8_t {
                    // return pixel ? 255 : 0;
                    if (pixel == MASKED_PIXEL) {
                        return 0;
                    } else {  // if (pixel == VALID_PIXEL)
                        return 255;
                    }
                };

                // Usage in your existing code
                save_device_data_to_png(
                  d_dispersion_mask.get(),          // Device pointer to the 2D array
                  d_dispersion_mask.pitch_bytes(),  // Device pitch in bytes
                  width,                            // Width of the image
                  height,                           // Height of the image
                  stream,                           // CUDA stream
                  "first_pass_dispersion_result",   // Output filename
                  convert_pixel                     // Pixel transformation function
                );
            }
            // Write to TXT
            {
                auto is_valid_pixel = [](uint8_t pixel) { return pixel != 0; };

                save_device_data_to_txt(
                  d_dispersion_mask.get(),          // Device pointer to the 2D array
                  d_dispersion_mask.pitch_bytes(),  // Device pitch in bytes
                  width,                            // Width of the image
                  height,                           // Height of the image
                  stream,                           // CUDA stream
                  "first_pass_dispersion_result",   // Output filename
                  is_valid_pixel                    // Pixel condition function
                );
            }
        }
    }

    /*
     * Erosion pass
     * Erode the first pass results.
     * The surviving pixels are then used as a mask to exclude them
     * from the background calculation in the second pass.
    */
    PitchedMalloc<uint8_t> d_erosion_mask(width, height);
    {
        dim3 threads_per_erosion_block(32, 32);
        dim3 erosion_blocks(
          (width + threads_per_erosion_block.x - 1) / threads_per_erosion_block.x,
          (height + threads_per_erosion_block.y - 1) / threads_per_erosion_block.y);

        // Calculate the shared memory size for the erosion kernel
        size_t erosion_shared_memory =
          (threads_per_erosion_block.x + 2 * first_pass_kernel_radius)
          * (threads_per_erosion_block.y + 2 * first_pass_kernel_radius)
          * sizeof(uint8_t);

        // Perform erosion
        erosion_kernel<<<erosion_blocks,
                         threads_per_erosion_block,
                         erosion_shared_memory,
                         stream>>>(d_dispersion_mask.get(),
                                   d_erosion_mask.get(),
                                   mask.get(),
                                   d_dispersion_mask.pitch,
                                   d_erosion_mask.pitch,
                                   mask.pitch,
                                   width,
                                   height,
                                   first_pass_kernel_radius);
        hipStreamSynchronize(stream);

        // Print the erosion result if needed
        if (do_writeout) {
            // Write to PNG
            {
                auto show_masked = [](uint8_t pixel) -> uint8_t {
                    if (pixel == MASKED_PIXEL) {
                        return 0;
                    } else {  // if (pixel == VALID_PIXEL)
                        return 255;
                    }
                };

                save_device_data_to_png(
                  d_erosion_mask.get(),          // Device pointer to the 2D array
                  d_erosion_mask.pitch_bytes(),  // Device pitch in bytes
                  width,                         // Width of the image
                  height,                        // Height of the image
                  stream,                        // CUDA stream
                  "eroded_dispersion_result",    // Output filename
                  show_masked                    // Pixel transformation function
                );
            }
            // Write to TXT
            {
                auto is_masked_pixel = [](uint8_t pixel) {
                    return pixel == MASKED_PIXEL;
                };

                save_device_data_to_txt(
                  d_erosion_mask.get(),          // Device pointer to the 2D array
                  d_erosion_mask.pitch_bytes(),  // Device pitch in bytes
                  width,                         // Width of the image
                  height,                        // Height of the image
                  stream,                        // CUDA stream
                  "eroded_dispersion_result",    // Output filename
                  is_masked_pixel                // Pixel condition function
                );
            }
        }
    }

    constexpr uint8_t second_pass_kernel_radius = 5;

    /*
     * Second pass
     * Perform the final thresholding using the dispersion mask.
    */
    {
        printf("Second pass\n");
        // Second pass: Perform the final thresholding using the dispersion mask
        compute_final_threshold_kernel<<<blocks, threads, shared_memory, stream>>>(
          image.get(),                // Image data pointer
          mask.get(),                 // Mask data pointer
          d_erosion_mask.get(),       // Dispersion mask pointer
          result_strong->get(),       // Output result mask pointer
          image.pitch,                // Image pitch
          mask.pitch,                 // Mask pitch
          d_erosion_mask.pitch,       // Dispersion mask pitch
          result_strong->pitch,       // Output result mask pitch
          width,                      // Image width
          height,                     // Image height
          max_valid_pixel_value,      // Maximum valid pixel value
          second_pass_kernel_radius,  // Kernel radius
          second_pass_kernel_radius,  // Kernel radius
          n_sig_s,                    // Signal significance level
          threshold                   // Global threshold
        );
        hipStreamSynchronize(
          stream);  // Synchronize the CUDA stream to ensure the second pass is complete

        printf("Second pass complete\n");
        // Optional: Write out the final result if needed
        if (do_writeout) {
            auto convert_pixel = [](uint8_t pixel) -> uint8_t {
                if (pixel == VALID_PIXEL) {
                    return 255;
                } else {
                    return 0;
                }
            };

            save_device_data_to_png(
              result_strong->get(),               // Device pointer to the 2D array
              mask.pitch_bytes(),                 // Device pitch in bytes
              width,                              // Width of the image
              height,                             // Height of the image
              stream,                             // CUDA stream
              "final_extended_threshold_result",  // Output filename
              convert_pixel                       // Pixel transformation function
            );

            auto is_valid_pixel = [](uint8_t pixel) { return pixel != 0; };

            save_device_data_to_txt(
              result_strong->get(),               // Device pointer to the 2D array
              mask.pitch_bytes(),                 // Device pitch in bytes
              width,                              // Width of the image
              height,                             // Height of the image
              stream,                             // CUDA stream
              "final_extended_threshold_result",  // Output filename
              is_valid_pixel                      // Pixel condition function
            );
        }
    }
}

#pragma endregion Launch Wrappers
