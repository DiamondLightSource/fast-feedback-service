#include "hip/hip_runtime.h"
/**
 * Basic Naive Kernel
 * 
 * Does spotfinding in-kernel, without in-depth performance tweaking.
 * 
 */

// #include <bitshuffle.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "spotfinder.h"

namespace cg = cooperative_groups;

__global__ void do_spotfinding_naive(pixel_t *image,
                                     size_t image_pitch,
                                     uint8_t *mask,
                                     size_t mask_pitch,
                                     int width,
                                     int height,
                                     //  int *result_sum,
                                     //  size_t *result_sumsq,
                                     //  uint8_t *result_n,
                                     uint8_t *result_strong) {
    image = image + (image_pitch * height * blockIdx.z);
    // result_sum = result_sum + (image_pitch * height * blockIdx.z);
    // result_sumsq = result_sumsq + (image_pitch * height * blockIdx.z);
    // result_n = result_n + (mask_pitch * height * blockIdx.z);
    result_strong = result_strong + (mask_pitch * height * blockIdx.z);

    auto block = cg::this_thread_block();
    // auto warp = cg::tiled_partition<32>(block);
    // int warpId = warp.meta_group_rank();
    // int lane = warp.thread_rank();

    uint sum = 0;
    size_t sumsq = 0;
    uint8_t n = 0;

    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    // Don't calculate for masked pixels
    bool px_is_valid = mask[y * mask_pitch + x] != 0;
    pixel_t this_pixel = image[y * image_pitch + x];

    if (px_is_valid) {
        for (int row = max(0, y - KERNEL_HEIGHT);
             row < min(y + KERNEL_HEIGHT + 1, height);
             ++row) {
            int row_offset = image_pitch * row;
            int mask_offset = mask_pitch * row;
            for (int col = max(0, x - KERNEL_WIDTH);
                 col < min(x + KERNEL_WIDTH + 1, width);
                 ++col) {
                pixel_t pixel = image[row_offset + col];
                uint8_t mask_pixel = mask[mask_offset + col];
                if (mask_pixel) {
                    sum += pixel;
                    sumsq += pixel * pixel;
                    n += 1;
                }
            }
        }
    }

    if (x < width && y < height) {
        // result_sum[x + image_pitch * y] = sum;
        // result_sumsq[x + image_pitch * y] = sumsq;
        // result_n[x + mask_pitch * y] = n;

        // Calculate the thresholding
        if (px_is_valid) {
            constexpr float n_sig_s = 3.0f;
            constexpr float n_sig_b = 6.0f;

            float sum_f = static_cast<float>(sum);
            float sumsq_f = static_cast<float>(sumsq);

            float mean = sum_f / n;
            float variance = (n * sumsq_f - (sum_f * sum_f)) / (n * (n - 1));
            float dispersion = variance / mean;
            float background_threshold = 1 + n_sig_b * sqrt(2.0f / (n - 1));
            bool not_background = dispersion > background_threshold;
            float signal_threshold = mean + n_sig_s * sqrt(mean);
            bool is_signal = this_pixel > signal_threshold;
            bool is_strong_pixel = not_background && is_signal;
            result_strong[x + mask_pitch * y] = is_strong_pixel;
        } else {
            result_strong[x + mask_pitch * y] = 0;
        }
    }
}
void call_do_spotfinding_naive(dim3 blocks,
                               dim3 threads,
                               size_t shared_memory,
                               hipStream_t stream,
                               pixel_t *image,
                               size_t image_pitch,
                               uint8_t *mask,
                               size_t mask_pitch,
                               int width,
                               int height,
                               //  int *result_sum,
                               //  size_t *result_sumsq,
                               //  uint8_t *result_n,
                               uint8_t *result_strong) {
    do_spotfinding_naive<<<blocks, threads, shared_memory, stream>>>(
      image, image_pitch, mask, mask_pitch, width, height, result_strong);
}
