#include "hip/hip_runtime.h"
/**
 * Basic Naive Kernel
 * 
 * Does spotfinding in-kernel, without in-depth performance tweaking.
 * 
 */

// #include <bitshuffle.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <lodepng.h>

#include "kernels/erosion.cuh"
#include "spotfinder.cuh"

namespace cg = cooperative_groups;

#pragma region Res Mask Functions
/**
 * @brief Function to calculate the distance of a pixel from the beam center.
 * @param x The x-coordinate of the pixel in the image
 * @param y The y-coordinate of the pixel in the image
 * @param center_x The x-coordinate of the pixel beam center in the image
 * @param center_y The y-coordinate of the pixel beam center in the image
 * @param pixel_size_x The pixel size of the detector in the x-direction in m
 * @param pixel_size_y The pixel size of the detector in the y-direction in m
 * @return The calculated distance from the beam center in m
*/
__device__ float get_distance_from_centre(float x,
                                          float y,
                                          float centre_x,
                                          float centre_y,
                                          float pixel_size_x,
                                          float pixel_size_y) {
    /*
     * Since this calculation is for a broad, general exclusion, we can
     * use basic Pythagoras to calculate the distance from the center.
    */
    // float dx = (x - centre_x) * pixel_size_x;
    // float dy = (y - centre_y) * pixel_size_y;

    /*
     * TODO: Check if the calculation should be done from the center of the pixel
     * or the corner of the pixel. The current calculation is from the center.
     * If the calculation should be from the corner, the calculation should be:
    */
    float dx = ((x + 0.5f) - centre_x) * pixel_size_x;
    float dy = ((y + 0.5f) - centre_y) * pixel_size_y;
    return sqrtf(dx * dx + dy * dy);
}

/**
 * @brief Function to calculate the interplanar distance of a reflection.
 * The interplanar distance is calculated using the formula:
 *         d = λ / (2 * sin(ϴ))
 * @param wavelength The wavelength of the X-ray beam in Å
 * @param distance_to_detector The distance from the sample to the detector in m
 * @param distance_from_center The distance of the reflection from the beam center in m
 * @return The calculated d value
*/
__device__ float get_resolution(float wavelength,
                                float distance_to_detector,
                                float distance_from_centre) {
    /*
     * Since the angle calculated is, in fact, 2ϴ, we halve to get the
     * proper value of ϴ
    */
    float theta = 0.5 * atanf(distance_from_centre / distance_to_detector);
    return wavelength / (2 * sinf(theta));
}
#pragma endregion Res Mask Functions

#pragma region Res Mask Kernel
/**
 * @brief CUDA kernel to apply a resolution mask for an image.
 *
 * This kernel calculates the resolution for each pixel in an image based on the
 * distance from the beam center and the detector properties. It then masks out
 * pixels whose resolution falls outside the specified range [dmin, dmax],
 * provided that the pixel is not already masked, by setting the mask value of
 * the pixel to 0 in the mask data.
 *
 * @param mask Pointer to the mask data indicating valid pixels.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param wavelength The wavelength of the X-ray beam in Ångströms.
 * @param distance_to_detector The distance from the sample to the detector in m.
 * @param beam_center_x The x-coordinate of the beam center in the image.
 * @param beam_center_y The y-coordinate of the beam center in the image.
 * @param pixel_size_x The pixel size of the detector in the x-direction in m.
 * @param pixel_size_y The pixel size of the detector in the y-direction in m.
 * @param dmin The minimum resolution (d-spacing) threshold.
 * @param dmax The maximum resolution (d-spacing) threshold.
 */
__global__ void apply_resolution_mask(uint8_t *mask,
                                      size_t mask_pitch,
                                      int width,
                                      int height,
                                      float wavelength,
                                      float distance_to_detector,
                                      float beam_center_x,
                                      float beam_center_y,
                                      float pixel_size_x,
                                      float pixel_size_y,
                                      float dmin,
                                      float dmax) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > width || y > height) return;  // Out of bounds

    if (mask[y * mask_pitch + x] == MASKED_PIXEL) {  // Check if the pixel is masked
        /*
        * If the pixel is already masked, we don't need to calculate the
        * resolution for it, so we can just leave it masked
        */
        return;
    }

    float distance_from_centre = get_distance_from_centre(
      x, y, beam_center_x, beam_center_y, pixel_size_x, pixel_size_y);
    float resolution =
      get_resolution(wavelength, distance_to_detector, distance_from_centre);

    // Check if dmin is set and if the resolution is below it
    if (dmin > 0 && resolution < dmin) {
        mask[y * mask_pitch + x] = MASKED_PIXEL;
        return;
    }

    // Check if dmax is set and if the resolution is above it
    if (dmax > 0 && resolution > dmax) {
        mask[y * mask_pitch + x] = MASKED_PIXEL;
        return;
    }

    // If the pixel is not masked and the resolution is within the limits, set the resolution mask to 1
    mask[y * mask_pitch + x] = VALID_PIXEL;
    // ⛔🧊
}

/**
 * @brief Host function to launch the apply_resolution_mask kernel.
 *
 * This function sets up the kernel execution parameters and launches the
 * apply_resolution_mask kernel to generate and apply a resolution mask
 * onto the base mask for the detector.
 *
 * @param blocks The dimensions of the grid of blocks.
 * @param threads The dimensions of the grid of threads within each block.
 * @param shared_memory The size of shared memory required per block (in bytes).
 * @param stream The CUDA stream to execute the kernel.
 * @param mask Device pointer to the mask data indicating valid pixels.
 * @param params The parameters required to calculate the resolution mask.  
 */
void call_apply_resolution_mask(dim3 blocks,
                                dim3 threads,
                                size_t shared_memory,
                                hipStream_t stream,
                                uint8_t *mask,
                                ResolutionMaskParams params) {
    // Launch the kernel
    apply_resolution_mask<<<blocks, threads, shared_memory, stream>>>(
      mask,
      params.mask_pitch,
      params.width,
      params.height,
      params.wavelength,
      params.detector.distance,
      params.detector.beam_center_x,
      params.detector.beam_center_y,
      params.detector.pixel_size_x,
      params.detector.pixel_size_y,
      params.dmin,
      params.dmax);
}
#pragma endregion Res Mask Kernel

#pragma region Spotfinding Functions
/**
 * @brief Calculate the sum, sum of squares, and count of valid pixels in the neighborhood.
 * @param image Device pointer to the image data.
 * @param mask Device pointer to the mask data indicating valid pixels.
 * @param background_mask (Optional) Device pointer to the background mask data. If nullptr, all pixels are considered for background calculation.
 * @param image_pitch The pitch (width in bytes) of the image data.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param x The x-coordinate of the current pixel.
 * @param y The y-coordinate of the current pixel.
 * @param kernel_width The radius of the kernel in the x-direction.
 * @param kernel_height The radius of the kernel in the y-direction.
 * @param sum (Output) The sum of the valid pixels in the neighborhood.
 * @param sumsq (Output) The sum of squares of the valid pixels in the neighborhood.
 * @param n (Output) The count of valid pixels in the neighborhood.
 */
__device__ void calculate_sums(pixel_t *image,
                               uint8_t *mask,
                               uint8_t *background_mask,
                               size_t image_pitch,
                               size_t mask_pitch,
                               int width,
                               int height,
                               int x,
                               int y,
                               int kernel_width,
                               int kernel_height,
                               uint &sum,
                               size_t &sumsq,
                               uint8_t &n) {
    sum = 0;
    sumsq = 0;
    n = 0;

    for (int row = max(0, y - kernel_height); row < min(y + kernel_height + 1, height);
         ++row) {
        int row_offset = image_pitch * row;
        int mask_offset = mask_pitch * row;
        for (int col = max(0, x - kernel_width); col < min(x + kernel_width + 1, width);
             ++col) {
            pixel_t pixel = image[row_offset + col];
            uint8_t mask_pixel = mask[mask_offset + col];
            bool include_pixel = mask_pixel != 0;  // If the pixel is valid
            if (background_mask != nullptr) {
                uint8_t background_mask_pixel = background_mask[mask_offset + col];
                include_pixel =
                  include_pixel
                  && (background_mask_pixel
                      == VALID_PIXEL);  // And is NOT a survivor from the erosion process
            }
            if (include_pixel) {
                sum += pixel;
                sumsq += pixel * pixel;
                n += 1;
            }
        }
    }
}

/**
 * @brief Determine if the current pixel is a strong pixel.
 * @param sum The sum of the valid pixels in the neighborhood.
 * @param sumsq The sum of squares of the valid pixels in the neighborhood.
 * @param n The count of valid pixels in the neighborhood.
 * @param this_pixel The intensity value of the current pixel.
 * @return True if the current pixel is a strong pixel, false otherwise.
 */
__device__ bool is_strong_pixel(uint sum, size_t sumsq, uint8_t n, pixel_t this_pixel) {
    constexpr float n_sig_s = 3.0f;
    constexpr float n_sig_b = 6.0f;

    float sum_f = static_cast<float>(sum);
    float sumsq_f = static_cast<float>(sumsq);

    float mean = sum_f / n;
    float variance = (n * sumsq_f - (sum_f * sum_f)) / (n * (n - 1));
    float dispersion = variance / mean;
    float background_threshold = 1 + n_sig_b * sqrt(2.0f / (n - 1));
    bool not_background = dispersion > background_threshold;
    float signal_threshold = mean + n_sig_s * sqrt(mean);
    bool is_signal = this_pixel > signal_threshold;

    return not_background && is_signal;
}
#pragma endregion Spotfinding Functions

#pragma region Spotfinding Kernels
/**
 * @brief CUDA kernel to perform spotfinding using a dispersion-based algorithm.
 * 
 * This kernel identifies strong pixels in the image based on analysis of the pixel neighborhood.
 * 
 * @param image Device pointer to the image data.
 * @param image_pitch The pitch (width in bytes) of the image data.
 * @param mask Device pointer to the mask data indicating valid pixels.
 * @param background_mask (Optional) Device pointer to the background mask data. If nullptr, all pixels are considered for background calculation.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param kernel_width The radius of the kernel in the x-direction.
 * @param kernel_height The radius of the kernel in the y-direction.
 * @param result_strong (Output) Device pointer for the strong pixel mask data to be written to.
 */
__global__ void do_spotfinding_dispersion(pixel_t *image,
                                          size_t image_pitch,
                                          uint8_t *mask,
                                          uint8_t *background_mask,
                                          size_t mask_pitch,
                                          int width,
                                          int height,
                                          pixel_t max_valid_pixel_value,
                                          int kernel_width,
                                          int kernel_height,
                                          uint8_t *result_strong) {
    image = image + (image_pitch * height * blockIdx.z);
    // result_sum = result_sum + (image_pitch * height * blockIdx.z);
    // result_sumsq = result_sumsq + (image_pitch * height * blockIdx.z);
    // result_n = result_n + (mask_pitch * height * blockIdx.z);
    result_strong = result_strong + (mask_pitch * height * blockIdx.z);

    auto block = cg::this_thread_block();
    // auto warp = cg::tiled_partition<32>(block);
    // int warpId = warp.meta_group_rank();
    // int lane = warp.thread_rank();

    uint sum = 0;
    size_t sumsq = 0;
    uint8_t n = 0;

    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    // Don't calculate for masked pixels
    pixel_t this_pixel = image[y * image_pitch + x];
    bool px_is_valid =
      mask[y * mask_pitch + x] != 0 && this_pixel <= max_valid_pixel_value;

    if (px_is_valid) {
        calculate_sums(image,
                       mask,
                       background_mask,
                       image_pitch,
                       mask_pitch,
                       width,
                       height,
                       x,
                       y,
                       kernel_width,
                       kernel_height,
                       sum,
                       sumsq,
                       n);
    }

    if (x < width && y < height) {
        // result_sum[x + image_pitch * y] = sum;
        // result_sumsq[x + image_pitch * y] = sumsq;
        // result_n[x + mask_pitch * y] = n;

        // Calculate the thresholding
        if (px_is_valid && n > 1) {
            bool is_strong_pixel_flag = is_strong_pixel(sum, sumsq, n, this_pixel);
            result_strong[x + mask_pitch * y] = is_strong_pixel_flag;
        } else {
            result_strong[x + mask_pitch * y] = 0;
        }
    }
}
/**
 * @brief Kernel for computing the basic threshold based on variance and mean.
 * @param image Pointer to the input image data.
 * @param mask Pointer to the mask data indicating valid pixels.
 * @param result_mask Pointer to the output mask data where results will be stored.
 * @param image_pitch The pitch (width in bytes) of the image data.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param min_count Minimum number of valid pixels required to be considered a valid spot.
 * @param threshold The global threshold for intensity.
 * @param n_sig_b Background noise significance level.
 * @param n_sig_s Signal significance level.
 */
__global__ void compute_threshold_kernel(pixel_t *image,
                                         uint8_t *mask,
                                         uint8_t *result_mask,
                                         size_t image_pitch,
                                         size_t mask_pitch,
                                         size_t result_pitch,
                                         int width,
                                         int height,
                                         pixel_t max_valid_pixel_value,
                                         int kernel_width,
                                         int kernel_height,
                                         int min_count,
                                         float n_sig_b,
                                         float n_sig_s) {
    // Move pointers to the correct slice
    image = image + (image_pitch * height * blockIdx.z);
    result_mask = result_mask + (mask_pitch * height * blockIdx.z);

    // Calculate the pixel coordinates
    auto block = cg::this_thread_block();
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    if (x >= width || y >= height) return;  // Out of bounds guard

    pixel_t this_pixel = image[y * image_pitch + x];

    // Check if the pixel is masked and below the maximum valid pixel value
    bool px_is_valid =
      mask[y * mask_pitch + x] != 0 && this_pixel <= max_valid_pixel_value;

    // Initialize variables for computing the local sum and count
    uint sum = 0;
    size_t sumsq = 0;
    uint8_t n = 0;

    int row_start = max(0, y - kernel_height);
    int row_end = min(y + kernel_height + 1, height);

    for (int row = row_start; row < row_end; ++row) {
        int row_offset = image_pitch * row;
        int mask_offset = mask_pitch * row;

        int col_start = max(0, x - kernel_width);
        int col_end = min(x + kernel_width + 1, width);

        for (int col = col_start; col < col_end; ++col) {
            pixel_t pixel = image[row_offset + col];
            uint8_t mask_pixel = mask[mask_offset + col];
            bool include_pixel = mask_pixel != 0;  // If the pixel is valid
            if (include_pixel) {
                sum += pixel;
                sumsq += pixel * pixel;
                n += 1;
            }
        }
    }

    if (px_is_valid && n > 1) {
        // Compute local mean and variance
        float sum_f = static_cast<float>(sum);
        float sumsq_f = static_cast<float>(sumsq);

        float mean = sum_f / n;
        float variance = (n * sumsq_f - (sum_f * sum_f)) / (n * (n - 1));
        float dispersion = variance / mean;

        // Compute the background threshold and signal threshold
        float background_threshold = 1 + n_sig_b * sqrt(2.0f / (n - 1));
        bool not_background = dispersion > background_threshold;
        float signal_threshold = mean + n_sig_s * sqrt(mean);

        // Check if the pixel is a strong pixel
        bool is_signal = this_pixel > signal_threshold;

        result_mask[x + result_pitch * y] = not_background && is_signal;
    } else {
        result_mask[x + result_pitch * y] = 0;
    }
}

/**
 * @brief Kernel for computing the dispersion threshold.
 * @param image Pointer to the input image data.
 * @param mask Pointer to the mask data indicating valid pixels.
 * @param result_mask Pointer to the output mask data where results will be stored.
 * @param image_pitch The pitch (width in bytes) of the image data.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param kernel_width The radius of the kernel in the x-direction.
 * @param kernel_height The radius of the kernel in the y-direction.
 * @param min_count Minimum number of valid pixels required to be considered a valid spot.
 * @param n_sig_b Background noise significance level.
 * @param n_sig_s Signal significance level.
 */
__global__ void compute_dispersion_threshold_kernel(pixel_t *image,
                                                    uint8_t *mask,
                                                    uint8_t *result_mask,
                                                    size_t image_pitch,
                                                    size_t mask_pitch,
                                                    size_t result_pitch,
                                                    int width,
                                                    int height,
                                                    pixel_t max_valid_pixel_value,
                                                    int kernel_width,
                                                    int kernel_height,
                                                    int min_count,
                                                    float n_sig_b,
                                                    float n_sig_s) {
    // Move pointers to the correct slice
    image = image + (image_pitch * height * blockIdx.z);
    result_mask = result_mask + (mask_pitch * height * blockIdx.z);

    // Calculate the pixel coordinates
    auto block = cg::this_thread_block();
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    if (x >= width || y >= height) return;  // Out of bounds guard

    // if it is the first pixel in the image print the kernel width and height
    if (x == 0 && y == 0) {
        printf("Kernel width: %d, Kernel height: %d\n", kernel_width, kernel_height);
        printf("Block Idx.z: %d\n", blockIdx.z);
    }

    pixel_t this_pixel = image[y * image_pitch + x];

    // Check if the pixel is masked and below the maximum valid pixel value
    bool px_is_valid =
      mask[y * mask_pitch + x] != 0 && this_pixel <= max_valid_pixel_value;

    // Initialize variables for computing the local sum and count
    uint sum = 0;
    size_t sumsq = 0;
    uint8_t n = 0;

    int row_start = max(0, y - kernel_height);
    int row_end = min(y + kernel_height + 1, height);

    for (int row = row_start; row < row_end; ++row) {
        int row_offset = image_pitch * row;
        int mask_offset = mask_pitch * row;

        int col_start = max(0, x - kernel_width);
        int col_end = min(x + kernel_width + 1, width);

        for (int col = col_start; col < col_end; ++col) {
            pixel_t pixel = image[row_offset + col];
            uint8_t mask_pixel = mask[mask_offset + col];
            bool include_pixel = mask_pixel != 0;  // If the pixel is valid
            if (include_pixel) {
                sum += pixel;
                sumsq += pixel * pixel;
                n += 1;
            }
        }
    }

    // Calculate the thresholding
    if (px_is_valid && n > 1) {
        // Compute local mean and variance
        float sum_f = static_cast<float>(sum);
        float sumsq_f = static_cast<float>(sumsq);

        float mean = sum_f / n;
        float variance = (n * sumsq_f - (sum_f * sum_f)) / (n * (n - 1));
        float dispersion = variance / mean;

        // Compute the background threshold
        float background_threshold = 1 + n_sig_b * sqrt(2.0f / (n - 1));
        bool not_background = dispersion > background_threshold;

        result_mask[x + result_pitch * y] = not_background;
    } else {
        result_mask[x + result_pitch * y] = 0;
    }
}

/**
 * @brief Kernel for computing the final threshold after dispersion mask.
 * @param image Pointer to the input image data.
 * @param mask Pointer to the mask data indicating valid pixels.
 * @param dispersion_mask Pointer to the dispersion mask used for extended algorithm.
 * @param result_mask Pointer to the output mask data where results will be stored.
 * @param image_pitch The pitch (width in bytes) of the image data.
 * @param mask_pitch The pitch (width in bytes) of the mask data.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param n_sig_s Signal significance level.
 * @param threshold Global threshold for the intensity.
 */
__global__ void compute_final_threshold_kernel(pixel_t *image,
                                               uint8_t *mask,
                                               uint8_t *dispersion_mask,
                                               uint8_t *result_mask,
                                               size_t image_pitch,
                                               size_t mask_pitch,
                                               size_t dispersion_mask_pitch,
                                               size_t result_mask_pitch,
                                               int width,
                                               int height,
                                               pixel_t max_valid_pixel_value,
                                               int kernel_width,
                                               int kernel_height,
                                               float n_sig_s,
                                               float threshold) {
    // Move pointers to the correct slice
    image = image + (image_pitch * height * blockIdx.z);
    result_mask = result_mask + (mask_pitch * height * blockIdx.z);

    // Calculate the pixel coordinates
    auto block = cg::this_thread_block();
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    if (x >= width || y >= height) return;  // Out of bounds guard

    pixel_t this_pixel = image[y * image_pitch + x];

    // Check if the pixel is masked and below the maximum valid pixel value
    bool px_is_valid =
      mask[y * mask_pitch + x] != 0 && this_pixel <= max_valid_pixel_value;

    // Initialize variables for computing the local sum and count
    uint sum = 0;
    uint8_t n = 0;

    int row_start = max(0, y - kernel_height);
    int row_end = min(y + kernel_height + 1, height);

    for (int row = row_start; row < row_end; ++row) {
        int row_offset = image_pitch * row;
        int mask_offset = mask_pitch * row;

        int col_start = max(0, x - kernel_width);
        int col_end = min(x + kernel_width + 1, width);

        for (int col = col_start; col < col_end; ++col) {
            pixel_t pixel = image[row_offset + col];
            uint8_t mask_pixel = mask[mask_offset + col];
            uint8_t disp_mask_pixel =
              dispersion_mask[row * dispersion_mask_pitch + col];
            bool include_pixel =
              mask_pixel != 0 && disp_mask_pixel;  // If the pixel is valid
            if (include_pixel) {
                sum += pixel;
                n += 1;
            }
        }
    }

    // Calculate the thresholding
    if (px_is_valid && n > 1) {
        float sum_f = static_cast<float>(sum);

        bool disp_mask = !dispersion_mask[y * dispersion_mask_pitch + x];
        bool global_mask = image[y * image_pitch + x] > threshold;
        float mean = sum_f / n;
        bool local_mask = image[y * image_pitch + x] >= (mean + n_sig_s * sqrtf(mean));

        result_mask[y * result_mask_pitch + x] = disp_mask && global_mask && local_mask;
    } else {
        result_mask[y * result_mask_pitch + x] = 0;
    }
}
#pragma endregion Spotfinding Kernel

#pragma region Launch Wrappers
/**
 * @brief Wrapper function to call the dispersion-based spotfinding algorithm.
 * This function launches the `compute_dispersion_threshold_kernel` to perform
 * the spotfinding based on the basic dispersion threshold.
 *
 * @param blocks The dimensions of the grid of blocks.
 * @param threads The dimensions of the grid of threads within each block.
 * @param shared_memory The size of shared memory required per block (in bytes).
 * @param stream The CUDA stream to execute the kernel.
 * @param image PitchedMalloc object for the image data.
 * @param mask PitchedMalloc object for the mask data indicating valid pixels.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param result_strong (Output) Device pointer for the strong pixel mask data to be written to.
 * @param min_count The minimum number of valid pixels required in the local neighborhood. Default is 3.
 * @param n_sig_b The background noise significance level. Default is 6.0.
 * @param n_sig_s The signal significance level. Default is 3.0.
 */
void call_do_spotfinding_dispersion(dim3 blocks,
                                    dim3 threads,
                                    size_t shared_memory,
                                    hipStream_t stream,
                                    PitchedMalloc<pixel_t> &image,
                                    PitchedMalloc<uint8_t> &mask,
                                    int width,
                                    int height,
                                    pixel_t max_valid_pixel_value,
                                    PitchedMalloc<uint8_t> *result_strong,
                                    int min_count,
                                    float n_sig_b,
                                    float n_sig_s) {
    /// One-direction width of kernel. Total kernel span is (K_W * 2 + 1)
    constexpr int basic_kernel_width = 3;
    /// One-direction height of kernel. Total kernel span is (K_H * 2 + 1)
    constexpr int basic_kernel_height = 3;

    // Launch the dispersion threshold kernel
    compute_threshold_kernel<<<blocks, threads, shared_memory, stream>>>(
      image.get(),            // Image data pointer
      mask.get(),             // Mask data pointer
      result_strong->get(),   // Output mask pointer
      image.pitch,            // Image pitch
      mask.pitch,             // Mask pitch
      result_strong->pitch,   // Output mask pitch
      width,                  // Image width
      height,                 // Image height
      max_valid_pixel_value,  // Maximum valid pixel value
      basic_kernel_width,     // Kernel width
      basic_kernel_height,    // Kernel height
      min_count,              // Minimum count
      n_sig_b,                // Background significance level
      n_sig_s                 // Signal significance level
    );

    // do_spotfinding_dispersion<<<blocks, threads, shared_memory, stream>>>(
    //   image.get(),
    //   image.pitch,
    //   mask.get(),
    //   nullptr,  // No background mask
    //   mask.pitch,
    //   width,
    //   height,
    //   max_valid_pixel_value,
    //   basic_kernel_width,
    //   basic_kernel_height,
    //   result_strong->get());

    hipStreamSynchronize(
      stream);  // Synchronize the CUDA stream to ensure the kernel is complete
}

/**
 * @brief Wrapper function to call the extended dispersion-based spotfinding algorithm.
 * This function launches the `compute_final_threshold_kernel` for final thresholding
 * after applying the dispersion mask and the `compute_dispersion_threshold_kernel`
 * for initial thresholding.
 *
 * @param blocks The dimensions of the grid of blocks.
 * @param threads The dimensions of the grid of threads within each block.
 * @param shared_memory The size of shared memory required per block (in bytes).
 * @param stream The CUDA stream to execute the kernel.
 * @param image PitchedMalloc object for the image data.
 * @param mask PitchedMalloc object for the mask data indicating valid pixels.
 * @param width The width of the image.
 * @param height The height of the image.
 * @param max_valid_pixel_value The maximum valid trusted pixel value.
 * @param result_strong (Output) Device pointer for the strong pixel mask data to be written to.
 * @param do_writeout Flag to indicate if the output should be written to file. Default is false.
 * @param min_count The minimum number of valid pixels required in the local neighborhood. Default is 3.
 * @param n_sig_b The background noise significance level. Default is 6.0.
 * @param n_sig_s The signal significance level. Default is 3.0.
 * @param threshold The global threshold for intensity values. Default is 10.0.
 */
void call_do_spotfinding_extended(dim3 blocks,
                                  dim3 threads,
                                  size_t shared_memory,
                                  hipStream_t stream,
                                  PitchedMalloc<pixel_t> &image,
                                  PitchedMalloc<uint8_t> &mask,
                                  int width,
                                  int height,
                                  pixel_t max_valid_pixel_value,
                                  PitchedMalloc<uint8_t> *result_strong,
                                  bool do_writeout,
                                  int min_count,
                                  float n_sig_b,
                                  float n_sig_s,
                                  float threshold) {
    // Allocate intermediate buffer for the dispersion mask on the device
    PitchedMalloc<uint8_t> d_dispersion_mask(width, height);

    constexpr int first_pass_kernel_radius = 3;

    /*
     * First pass
     * Perform the initial dispersion thresholding only on the background
     * threshold. The surviving pixels are then used as a mask later to
     * exclude them from the background calculation in the second pass.
    */
    {
        printf("First pass\n");
        // First pass: Perform the initial dispersion thresholding
        compute_dispersion_threshold_kernel<<<blocks, threads, shared_memory, stream>>>(
          image.get(),               // Image data pointer
          mask.get(),                // Mask data pointer
          d_dispersion_mask.get(),   // Output dispersion mask pointer
          image.pitch,               // Image pitch
          mask.pitch,                // Mask pitch
          d_dispersion_mask.pitch,   // Output dispersion mask pitch
          width,                     // Image width
          height,                    // Image height
          max_valid_pixel_value,     // Maximum valid pixel value
          first_pass_kernel_radius,  // Kernel radius
          first_pass_kernel_radius,  // Kernel radius
          min_count,                 // Minimum count
          n_sig_b,                   // Background significance level
          n_sig_s                    // Signal significance level
        );
        hipStreamSynchronize(
          stream);  // Synchronize the CUDA stream to ensure the first pass is complete

        printf("First pass complete\n");
        // Optional: Write out the first pass result if needed
        if (do_writeout) {
            // Write to PNG
            {
                // Function to transform the pixel values: if non-zero, set to 255, otherwise set to 0
                auto convert_pixel = [](uint8_t pixel) -> uint8_t {
                    // return pixel ? 255 : 0;
                    if (pixel == MASKED_PIXEL) {
                        return 0;
                    } else {  // if (pixel == VALID_PIXEL)
                        return 255;
                    }
                };

                // Usage in your existing code
                save_device_data_to_png(
                  d_dispersion_mask.get(),          // Device pointer to the 2D array
                  d_dispersion_mask.pitch_bytes(),  // Device pitch in bytes
                  width,                            // Width of the image
                  height,                           // Height of the image
                  stream,                           // CUDA stream
                  "first_pass_dispersion_result",   // Output filename
                  convert_pixel                     // Pixel transformation function
                );
            }
            // Write to TXT
            {
                auto is_valid_pixel = [](uint8_t pixel) { return pixel != 0; };

                save_device_data_to_txt(
                  d_dispersion_mask.get(),          // Device pointer to the 2D array
                  d_dispersion_mask.pitch_bytes(),  // Device pitch in bytes
                  width,                            // Width of the image
                  height,                           // Height of the image
                  stream,                           // CUDA stream
                  "first_pass_dispersion_result",   // Output filename
                  is_valid_pixel                    // Pixel condition function
                );
            }
        }
    }

    /*
     * Erosion pass
     * Erode the first pass results.
     * The surviving pixels are then used as a mask to exclude them
     * from the background calculation in the second pass.
    */
    {
        dim3 threads_per_erosion_block(32, 32);
        dim3 erosion_blocks(
          (width + threads_per_erosion_block.x - 1) / threads_per_erosion_block.x,
          (height + threads_per_erosion_block.y - 1) / threads_per_erosion_block.y);

        // Calculate the shared memory size for the erosion kernel
        size_t erosion_shared_memory =
          (threads_per_erosion_block.x + 2 * first_pass_kernel_radius)
          * (threads_per_erosion_block.y + 2 * first_pass_kernel_radius)
          * sizeof(uint8_t);

        // Perform erosion
        erosion_kernel<<<erosion_blocks,
                         threads_per_erosion_block,
                         erosion_shared_memory,
                         stream>>>(d_dispersion_mask.get(),
                                   d_dispersion_mask.pitch_bytes(),
                                   width,
                                   height,
                                   first_pass_kernel_radius);
        hipStreamSynchronize(stream);

        // Print the erosion result if needed
        if (do_writeout) {
            auto show_masked = [](uint8_t pixel) -> uint8_t {
                if (pixel == MASKED_PIXEL) {
                    return 255;
                } else {  // if (pixel == VALID_PIXEL)
                    return 0;
                }
            };

            save_device_data_to_png(
              d_dispersion_mask.get(),          // Device pointer to the 2D array
              d_dispersion_mask.pitch_bytes(),  // Device pitch in bytes
              width,                            // Width of the image
              height,                           // Height of the image
              stream,                           // CUDA stream
              "eroded_dispersion_result",       // Output filename
              show_masked                       // Pixel transformation function
            );
        }
    }

    constexpr int second_pass_kernel_radius = 5;

    /*
     * Second pass
     * Perform the final thresholding using the dispersion mask.
    */
    {
        printf("Second pass\n");
        // Second pass: Perform the final thresholding using the dispersion mask
        compute_final_threshold_kernel<<<blocks, threads, shared_memory, stream>>>(
          image.get(),                // Image data pointer
          mask.get(),                 // Mask data pointer
          d_dispersion_mask.get(),    // Dispersion mask pointer
          result_strong->get(),       // Output result mask pointer
          image.pitch,                // Image pitch
          mask.pitch,                 // Mask pitch
          d_dispersion_mask.pitch,    // Dispersion mask pitch
          result_strong->pitch,       // Output result mask pitch
          width,                      // Image width
          height,                     // Image height
          max_valid_pixel_value,      // Maximum valid pixel value
          second_pass_kernel_radius,  // Kernel radius
          second_pass_kernel_radius,  // Kernel radius
          n_sig_s,                    // Signal significance level
          threshold                   // Global threshold
        );
        hipStreamSynchronize(
          stream);  // Synchronize the CUDA stream to ensure the second pass is complete

        printf("Second pass complete\n");
        // Optional: Write out the final result if needed
        if (do_writeout) {
            auto convert_pixel = [](uint8_t pixel) -> uint8_t {
                if (pixel == VALID_PIXEL) {
                    return 255;
                } else {
                    return 0;
                }
            };

            save_device_data_to_png(
              result_strong->get(),               // Device pointer to the 2D array
              mask.pitch_bytes(),                 // Device pitch in bytes
              width,                              // Width of the image
              height,                             // Height of the image
              stream,                             // CUDA stream
              "final_extended_threshold_result",  // Output filename
              convert_pixel                       // Pixel transformation function
            );

            auto is_valid_pixel = [](uint8_t pixel) { return pixel != 0; };

            save_device_data_to_txt(
              result_strong->get(),               // Device pointer to the 2D array
              mask.pitch_bytes(),                 // Device pitch in bytes
              width,                              // Width of the image
              height,                             // Height of the image
              stream,                             // CUDA stream
              "final_extended_threshold_result",  // Output filename
              is_valid_pixel                      // Pixel condition function
            );
        }
    }
}

#pragma endregion Launch Wrappers
